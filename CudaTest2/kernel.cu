#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <Math.h>
#include <stdio.h>

hipError_t addWithCuda(double *sinx, unsigned int size);

__global__ void addKernel(double *sinx)
{
    //int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	sinx[i] = sin(i*3.14 / 1024);

}

int main()
{
    const int arraySize = 1024;
    double sinx[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(sinx, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	printf("sin[0;pi) with 1024 steps\n\n");


	for (int i = 0; i < arraySize; i++)
	{
		printf("sin[%d*pi/1024] = %f\n", i, sinx[i]);
	}


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *sinx, unsigned int size)
{
    double *dev_sinx = 0;
 
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_sinx, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }



    // Launch a kernel on the GPU with one thread for each element.

    addKernel<<<4, 256>>>(dev_sinx);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(sinx, dev_sinx, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_sinx);
    
    return cudaStatus;
}
